#include "hip/hip_runtime.h"
#include "kernel.h"

#include <cmath>
#include <iostream>  // This include fixes segfault due to uninitialized std::cout, even if the latter is not used

#include "gpu.h"
#include "skeleton.h"
#include "wrapper.h"


namespace kernel {

void print_cuda_error(hipError_t err) {
    if (err != hipSuccess) std::cout << hipGetErrorString(err) << std::endl;
}

template <class T>
struct raw_array {
    T *data;
    GPUd() T& operator[](std::size_t i) { return data[i]; }
    GPUd() const T& operator[](std::size_t i) const { return data[i]; }
};

__global__
void add(int N, wrapper::wrapper<raw_array, S, wrapper::layout::aos> w) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) w[i].y = w[i].x + w[i].y;
}

int run() {
    int N = 8;
    wrapper::wrapper<raw_array, S, wrapper::layout::aos> w;
    print_cuda_error(hipMallocManaged(&w.data.data, N * sizeof(S<wrapper::value>)));
    for (int i = 0; i < N; ++i) {
        S<wrapper::reference> r = w[i];
        r.setX(1);
        r.y = 2;
        r.point = {0.5 * i, 0.5 * i};
        r.identifier = 0.1 * i;
    }

    // int blockSize = 1;
    // int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<1, 1>>>(N, w);
    print_cuda_error(hipDeviceSynchronize());

    int maxError = 0;
    for (int i = 0; i < N; ++i) maxError = std::max(maxError, std::abs(w[i].y - 3));
    print_cuda_error(hipFree(w.data.data));
    return maxError;
}

}  // namespace kernel

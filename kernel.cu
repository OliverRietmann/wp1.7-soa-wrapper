#include "hip/hip_runtime.h"
#include "kernel.h"

#include <iostream>  // This include fixes segfault due to uninitialized std::cout, even if the latter is not used

#include "gpu.h"
#include "skeleton.h"  // Needed only for forward declarations
#include "wrapper.h"

// #include <cuda/std/span>  // Should work out of the box


namespace kernel {

template <class T>
using pointer_type = T*;

int cuda_malloc_managed(void** data, std::size_t size) { return hipMallocManaged(data, size); }

int cuda_free(void* ptr) { return hipFree(ptr); }

void print_cuda_error(hipError_t err) {
    if (err != hipSuccess) std::cout << hipGetErrorString(err) << std::endl;
}

template <
    template <class> class F,
    template <template <class> class> class S,
    wrapper::layout L
>
__global__ void add(int N, wrapper::wrapper<F, S, L> w) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) w[i].y = w[i].getX() + w[i].y;
}

template <
    template <class> class F,
    template <template <class> class> class S,
    wrapper::layout L
>
void apply(int N, wrapper::wrapper<F, S, L> w) {
    add<<<1, 1>>>(N, w);
    print_cuda_error(hipDeviceSynchronize());
}

// Explicit instatiations needed for unit tests (TODO: Get rid of this)
template void apply<pointer_type, S, wrapper::layout::aos>(int N, wrapper::wrapper<pointer_type, S, wrapper::layout::aos> w);
template void apply<pointer_type, S, wrapper::layout::soa>(int N, wrapper::wrapper<pointer_type, S, wrapper::layout::soa> w);

}  // namespace kernel
